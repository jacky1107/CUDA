#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define N 10000000

__global__ void add(int *a, int *b, int *c)
{
    int tid = blockIdx.x;
    while (tid < N)
    {
        c[tid] = a[tid] + b[tid];
        tid += gridDim.x;
    }
}

int main(void)
{
    int *a, *b, *c;
    int *dev_a, *dev_b, *dev_c;

    float elapsedTime;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    a = (int *)malloc(N * sizeof(int));
    b = (int *)malloc(N * sizeof(int));
    c = (int *)malloc(N * sizeof(int));

    if (hipMalloc((void **)&dev_a, N * sizeof(int)) != hipSuccess) return 1;
    if (hipMalloc((void **)&dev_b, N * sizeof(int)) != hipSuccess) return 1 ;
    if (hipMalloc((void **)&dev_c, N * sizeof(int)) != hipSuccess) return 1 ;
    srand(time(NULL));
    for (int i = 0; i < N; i++)
    {
        a[i] = rand() % N;
        b[i] = rand() % N;
    }

    if (hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice) != hipSuccess) return 1 ;
    if (hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice) != hipSuccess) return 1 ;
    
    int per_blocks = N / (1024 * 2);
    printf("Per blocks: %d\n", per_blocks);

    hipEventRecord(start, 0);
    add<<<per_blocks, 1>>>(dev_a, dev_b, dev_c);
    hipEventRecord(stop, 0);
    
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("GPU time: %13f msec\n", elapsedTime);

    if (hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost) != hipSuccess) return 1 ;
    
    bool success = true;
    for (int i = 0; i < N; i++)
        if ((a[i] + b[i]) != c[i])
        {
            success = false;
            break;
        }
    if (success)
        printf("We did it!\n");
    else
        printf("Failed\n");

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}