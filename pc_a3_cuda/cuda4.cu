#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define NUM_ROWS 10000
#define NUM_COLS 1000

int a[NUM_ROWS][NUM_COLS];
int b[NUM_ROWS][NUM_COLS];
int c[NUM_ROWS][NUM_COLS];
int d[NUM_ROWS][NUM_COLS];

__global__ void add(int *a, int *b, int *c)
{
    int x = threadIdx.x;
    int y = threadIdx.y * blockDim.x;
    int tid = x + y;
    while (tid < NUM_ROWS * NUM_COLS) {
        c[tid] = a[tid] + b[tid];
        tid += blockDim.x * blockDim.y;
    }
}

int main(void)
{
    int *dev_a, *dev_b, *dev_c;
    int size = NUM_ROWS * NUM_COLS * sizeof(int);
    hipError_t cuError = hipSuccess ;

    float elapsedTime;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    if (hipMalloc((void **)&dev_a, size) != hipSuccess) return 1;
    if (hipMalloc((void **)&dev_b, size) != hipSuccess) return 1;
    if (hipMalloc((void **)&dev_c, size) != hipSuccess) return 1;
    for(int i = 0;i < NUM_ROWS; i++)
    {
        for(int j = 0;j < NUM_COLS; j++)
        {
            a[i][j] = rand() % NUM_ROWS + 1;
            b[i][j] = rand() % NUM_COLS + 1;
        }
    }

    int cpu = true;
    double elapsedTimeCPU;
    struct timespec t_start, t_end;
    if (cpu) {
        clock_gettime( CLOCK_REALTIME, &t_start);
        for(int i = 0;i < NUM_ROWS; i++)
        {
            for(int j = 0;j < NUM_COLS; j++)
            {
                d[i][j] = a[i][j] + b[i][j];
            }
        }
        clock_gettime( CLOCK_REALTIME, &t_end);
        elapsedTimeCPU = (t_end.tv_sec - t_start.tv_sec) * 1000.0;
        elapsedTimeCPU += (t_end.tv_nsec - t_start.tv_nsec) / 1000000.0;
        printf("CPU elapsedTime: %lf ms\n", elapsedTimeCPU);
    }

    if (hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice) != hipSuccess) return 1 ;
    if (hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice) != hipSuccess) return 1 ;
    
    dim3 dimGrid (1, 1, 1);
    dim3 dimBlock (32, 32, 1);

    hipEventRecord(start, 0);
    add<<<dimGrid, dimBlock>>>(dev_a, dev_b, dev_c);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("GPU time: %13f msec\n", elapsedTime);

    cuError = hipGetLastError();
    if (hipSuccess != cuError)
    {
        printf ("Failed in kernel launch and reason is %s\n", hipGetErrorString(cuError)) ;
        return 1 ;
    }
    
    if (hipMemcpy(c, dev_c, size, hipMemcpyDeviceToHost) != hipSuccess) return 1 ;

    bool success = true;
    for (int i = 0; i < NUM_ROWS; i++)
    {
        for (int j = 0; j < NUM_COLS; j++)
        {
            if ((a[i][j] + b[i][j]) != c[i][j])
            {
                success = false;
                break;
            }

        }
    }
    if (success)
        printf("We did it!\n");
    else
        printf("Failed\n");

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}