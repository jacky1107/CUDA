#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define NUM_ROWS 10000
#define NUM_COLS 1000

int a[NUM_ROWS][NUM_COLS];
int b[NUM_ROWS][NUM_COLS];
int c[NUM_ROWS][NUM_COLS];

__global__ void add(int *a, int *b, int *c)
{
    int x = blockIdx.x;
    int y = gridDim.x * blockIdx.y;
    int tid = x + y;
    while (tid < NUM_ROWS * NUM_COLS) {
        c[tid] = a[tid] + b[tid];
        tid += gridDim.x * gridDim.y;
    }
}

int main()
{
    int *dev_a, *dev_b, *dev_c;
    int size = NUM_ROWS * NUM_COLS * sizeof(int);
    hipError_t cuError = hipSuccess ;

    float elapsedTime;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    for(int i=0;i<NUM_ROWS;i++)
    {
        for(int j=0;j<NUM_COLS;j++)
        {
            a[i][j] = rand() % size + 1;
            b[i][j] = rand() % size + 1;
        }
    }
    if (hipMalloc((void **)&dev_a, size) != hipSuccess) return 1;
    if (hipMalloc((void **)&dev_b, size) != hipSuccess) return 1;
    if (hipMalloc((void **)&dev_c, size) != hipSuccess) return 1;
    if (hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice) != hipSuccess) return 1;
    if (hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice) != hipSuccess) return 1;

    int R = NUM_ROWS / 32;
    int C = NUM_COLS / 32;
    dim3 dimGrid (R, C, 1);
    dim3 dimBlock (1, 1, 1);

    hipEventRecord(start, 0);
    add<<<dimGrid, dimBlock>>>(dev_a, dev_b, dev_c);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("GPU time: %13f msec\n", elapsedTime);

    cuError = hipGetLastError();
    if (hipSuccess != cuError)
    {
        printf ("Failed in kernel launch and reason is %s\n", hipGetErrorString(cuError)) ;
        return 1 ;
    }
    if (hipMemcpy(c, dev_c, size, hipMemcpyDeviceToHost) != hipSuccess) return 1 ;

    bool success = true;
    for (int i = 0; i < NUM_ROWS; i++)
    {
        for (int j = 0; j < NUM_COLS; j++)
        {
            if ((a[i][j] + b[i][j]) != c[i][j])
            {
                success = false;
                break;
            }

        }
    }
    if (success)
        printf("We did it!\n");
    else
        printf("Failed\n");

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}