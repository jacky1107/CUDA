#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#define N 10000000
#define KN 5000
#define THREADSPERBLOCK 32
#define BLOCKSPERGRID 256

float data[N];
float kernel[KN];
float output[N-KN+1];
float output_from_device[N-KN+1];

__global__ void conv( float *data_cuda, float *kernel, float *output ){
    int tx = threadIdx.x;
    int ty = blockDim.x * threadIdx.y;
    int bx = blockDim.x * blockDim.y * blockIdx.x;
    int by = gridDim.x * (blockDim.x * blockDim.y) * blockIdx.y;
    int tid = bx + by + tx + ty;
    while (tid < N - KN + 1) {
        for(int i = 0; i < KN; i++) {
            output[tid] += data_cuda[tid + i] * kernel[i];
        }
        tid = tid + gridDim.x * gridDim.y * blockDim.x * blockDim.y;
    }
}

int main(){
    int cpu = true;
    int pass = 1;
    hipError_t cuError = hipSuccess ;

    double elapsedTimeCPU;
    struct timespec t_start, t_end;
    
    float elapsedTime;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // generate dummy data
    srand(time(NULL));
    for (int i = 0; i < KN; i++) {
        kernel[i] = rand() / (float)RAND_MAX;
    }
    
    srand(time(NULL));
    for (int i = 0; i < N; i++) {
        data[i] = rand() / (float)RAND_MAX;
    }

    // CPU
    if (cpu) {
        clock_gettime( CLOCK_REALTIME, &t_start);
        for (int i = 0; i < N-KN+1; i++) {
            output[i] = 0;
            for (int j = 0; j < KN; j++) {
                output[i] += kernel[j] * data[i+j];
            }
        }
        clock_gettime( CLOCK_REALTIME, &t_end);
        elapsedTimeCPU = (t_end.tv_sec - t_start.tv_sec) * 1000.0;
        elapsedTimeCPU += (t_end.tv_nsec - t_start.tv_nsec) / 1000000.0;
        printf("CPU elapsedTime: %lf ms\n", elapsedTimeCPU);
    }

    // GPU
    float *d_kernel, *d_data, *d_output;
    if (hipMalloc( (void**)&d_kernel, KN * sizeof(float) ) != hipSuccess) return 1;
    if (hipMalloc( (void**)&d_data, N * sizeof(float) ) != hipSuccess) return 1;
    if (hipMalloc( (void**)&d_output, (N-KN+1) * sizeof(float) ) != hipSuccess) return 1;
    if (hipMemcpy( d_kernel, kernel, KN * sizeof(float), hipMemcpyHostToDevice ) != hipSuccess) return 1;
    if (hipMemcpy( d_data, data, N * sizeof(float), hipMemcpyHostToDevice ) != hipSuccess) return 1;

    int per_threads_x = THREADSPERBLOCK;
    int per_threads_y = THREADSPERBLOCK;
    int per_blocks_x = BLOCKSPERGRID;
    int per_blocks_y = BLOCKSPERGRID;
    printf("%d %d\n", per_blocks_x, per_blocks_y);

    dim3 dimGrid (per_blocks_x, per_blocks_y, 1);
    dim3 dimBlock (per_threads_x, per_threads_y, 1);

    hipEventRecord(start, 0);
    conv<<<dimGrid, dimBlock>>>(d_data, d_kernel, d_output);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop); 
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("GPU time: %13f msec\n", elapsedTime);
    
    hipMemcpy( output_from_device, d_output, (N-KN+1) * sizeof(float), hipMemcpyDeviceToHost );
    hipEventDestroy(start);
    hipEventDestroy(stop);

    if (hipGetLastError() != hipSuccess)
    {
        printf ("Failed in kernel launch and reason is %s\n", hipGetErrorString(cuError)) ;
        return 1 ;
    }

    //check correctness
    if (cpu) {
        for (int i = 0; i < N-KN+1; i++){
            if((output_from_device[i] - output[i]) > 0.001){
                printf("CPU:%lf GPU:%lf\n",output[i], output_from_device[i] );
                pass = 0;
                break;
            }
        }
        if(pass == 1) {
            printf("Test pass!\n");
            printf("GPU / CPU = %f\n", elapsedTimeCPU / elapsedTime);
        }
        else
            printf("Test fail!\n");
    }
}