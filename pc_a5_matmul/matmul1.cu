#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#define WIDTH 256

__global__ void MatMulKernel(float *Md, float *Nd, float *Pd, int width);

int main(int argc, char *argv[])
{
    int width = WIDTH;
    float M[WIDTH][WIDTH] = {0};
    float N[WIDTH][WIDTH] = {0};
    float P[WIDTH][WIDTH] = {0};
    float MxN[WIDTH][WIDTH] = {0};
    int pass = 1;
    
    for (int i = 0; i < width; ++i) {
        for (int j = 0; j < width; ++j) {
            M[i][j] = rand() % 30;
            N[i][j] = rand() % 30;
        }
    }
    
    struct timeval starttime, endtime;
    gettimeofday(&starttime, NULL);
    for (int i = 0; i < width; ++i) {
        for (int j = 0; j < width; ++j) {
            for (int k = 0; k < width; ++k) {
                MxN[i][j] += M[i][k] * N[k][j];
            }
        }
    }
    gettimeofday(&endtime, NULL);
    double executime;
    executime = (endtime.tv_sec - starttime.tv_sec) * 1000.0;
    executime += (endtime.tv_usec - starttime.tv_usec) / 1000.0;
    printf("CPU time: %13lf msec\n", executime);
    
	size_t size = width * width * sizeof(float);
    float *Md, *Nd, *Pd;
    
    // Allocate and Load M, N to device memory
    hipMalloc((void **)&Md, size);
    hipMemcpy(Md, M, size, hipMemcpyHostToDevice);
    
    hipMalloc((void **)&Nd, size);
    hipMemcpy(Nd, N, size, hipMemcpyHostToDevice);
    
    // Allocate P on the device
    hipMalloc((void **)&Pd, size);
    
    // Setup the execution configuration
    dim3 dimGrid(1, 1);
    dim3 dimBlock(32, 32);
    
    // Get start time event
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    
    // Invoke kernel
    MatMulKernel<<<dimGrid, dimBlock>>>(Md, Nd, Pd, width);
    hipError_t cuda_err = hipGetLastError();
    if ( hipSuccess != cuda_err ){
        printf("before kernel call: error = %s\n", hipGetErrorString (cuda_err));
        exit(1) ;
    }
    
    // Get stop time event
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    // Compute execution time
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("GPU time: %13f msec\n", elapsedTime);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    // Read P from device memory
    hipMemcpy(P, Pd, size, hipMemcpyDeviceToHost);
    
    // Free device memory
    hipFree(Md);
    hipFree(Nd);
    hipFree(Pd);
    
    for (int i = 0; i < width; ++i) {
        for (int j = 0; j < width; ++j) {
            if((MxN[i][j] - P[i][j]) > 0.0001) {
				// printf("MxN[%d][%d] = %2.0f   P[%d][%d] = %2.0f\n", i, j, MxN[i][j], i, j, P[i][j]);
                pass = 0;
                break;
            }
        }
    }
    
    printf("Test %s\n", (pass)?"PASSED":"FAILED");
    
    return 0;
}

// Matrix multiplication kernel called by MatMul()
__global__ void MatMulKernel(float *Md, float *Nd, float *Pd, int width)
{
    int i = threadIdx.y;
    int j = threadIdx.x;
    int tid_i = i;
    int tid_j = j;
    while(tid_i < width) {
        while(tid_j < width) {
            float Pvalue = 0;
            for (int k = 0; k < width; ++k) {
                int x = tid_i * width + k;
                int y = k * width + tid_j;
                float Melement = *(Md + x);
                float Nelement = *(Nd + y);
                Pvalue += Melement * Nelement;
            }
            *(Pd + tid_i * width + tid_j) = Pvalue;
            tid_j += blockDim.x;
        }
        tid_i += blockDim.y;
        tid_j = threadIdx.x;
    }
}