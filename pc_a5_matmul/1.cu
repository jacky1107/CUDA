#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#define WIDTH 1024
#define THREADSPERBLOCK 16
#define BLOCKSPERGRID 1

int M[WIDTH][WIDTH] = {0};
int N[WIDTH][WIDTH] = {0};
int P[WIDTH][WIDTH] = {0};
int MxN[WIDTH][WIDTH] = {0};

__global__ void mat_mul(int *Md, int *Nd, int *Pd);

int main(int argc, char *argv[])
{
    float elapsedTime;

    for (int i = 0; i < WIDTH; ++i)
    {
        for (int j = 0; j < WIDTH; ++j)
        {
            M[i][j] = (int)(rand() % 255 + 1);
            N[i][j] = (int)(rand() % 255 + 1);
        }
    }

    struct timeval starttime, endtime;
    gettimeofday(&starttime, NULL);
    for (int i = 0; i < WIDTH; ++i)
    {
        for (int j = 0; j < WIDTH; ++j)
        {
            for (int k = 0; k < WIDTH; ++k)
            {
                MxN[i][j] += M[i][k] * N[k][j];
            }
        }
    }
    gettimeofday(&endtime, NULL);
    double executime;
    executime = (endtime.tv_sec - starttime.tv_sec) * 1000.0;
    executime += (endtime.tv_usec - starttime.tv_usec) / 1000.0;
    printf("CPU time: %13lf msec\n", executime);

    // Original
    size_t size = WIDTH * WIDTH * sizeof(int);
    int *Md, *Nd, *Pd;
    hipMalloc((void **)&Md, size);
    hipMemcpy(Md, M, size, hipMemcpyHostToDevice);
    hipMalloc((void **)&Nd, size);
    hipMemcpy(Nd, N, size, hipMemcpyHostToDevice);
    hipMalloc((void **)&Pd, size);

    dim3 dimGrid(BLOCKSPERGRID, BLOCKSPERGRID);
    dim3 dimBlock(THREADSPERBLOCK, THREADSPERBLOCK);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    mat_mul<<<dimGrid, dimBlock>>>(Md, Nd, Pd);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("GPU time: %13f msec\n", elapsedTime);
    hipError_t cuda_err = hipGetLastError();
    if (hipSuccess != cuda_err)
    {
        printf("before kernel call: error = %s\n", hipGetErrorString(cuda_err));
        exit(1);
    }
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipMemcpy(P, Pd, size, hipMemcpyDeviceToHost);
    int pass = 1;
    for (int i = 0; i < WIDTH; ++i)
    {
        for (int j = 0; j < WIDTH; ++j)
        {
            if (MxN[i][j] != P[i][j])
            {
                printf("MxN[%d][%d] = %d   P[%d][%d] = %d\n", i, j, MxN[i][j], i, j, P[i][j]);
                pass = 0;
                break;
            }
        }
    }
    printf("Test %s\n", (pass) ? "PASSED" : "FAILED");
    hipFree(Md);
    hipFree(Nd);
    hipFree(Pd);

    return 0;
}

__global__ void mat_mul(int *Md, int *Nd, int *Pd)
{
    int x, y;
    int Pvalue;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int tid = tx + ty * blockDim.x;

    while (tid < WIDTH * WIDTH)
    {
        x = tid % WIDTH;
        y = tid / WIDTH;

        Pvalue = 0;
        for (int k = 0; k < WIDTH; ++k)
        {
            int Melement = *(Md + y * WIDTH + k);
            int Nelement = *(Nd + k * WIDTH + x);
            Pvalue += Melement * Nelement;
        }

        *(Pd + y * WIDTH + x) = Pvalue;
        tid += blockDim.x * blockDim.y;
    }
}
