#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#define WIDTH 1024
#define TILE_WIDTH 16
#define BLOCKSPERGRID (WIDTH + TILE_WIDTH - 1) / TILE_WIDTH

int M[WIDTH][WIDTH] = {0};
int N[WIDTH][WIDTH] = {0};
int P[WIDTH][WIDTH] = {0};
int MxN[WIDTH][WIDTH] = {0};

__global__ void mat_mul(int *Md, int *Nd, int *Pd);
__global__ void transpose(int *Nd, int *Td);
__device__ int GetElement(int *matrix, int row, int col);
__device__ void SetElement(int *matrix, int row, int col, int value);
__device__ int *GetSubMatrix(int *matrix, int blockrow, int blockcol);

int main(int argc, char *argv[])
{
    float elapsedTime1;
    float elapsedTime2;
    
    for (int i = 0; i < WIDTH; ++i) {
        for (int j = 0; j < WIDTH; ++j) {
            M[i][j] = (int) (rand() % 255 + 1);
            N[i][j] = (int) (rand() % 255 + 1);
        }
    }

    struct timeval starttime, endtime;
    gettimeofday(&starttime, NULL);
    for (int i = 0; i < WIDTH; ++i) {
        for (int j = 0; j < WIDTH; ++j) {
            for (int k = 0; k < WIDTH; ++k) {
                MxN[i][j] += M[i][k] * N[k][j];
            }
        }
    }
    gettimeofday(&endtime, NULL);
    double executime;
    executime = (endtime.tv_sec - starttime.tv_sec) * 1000.0;
    executime += (endtime.tv_usec - starttime.tv_usec) / 1000.0;
    printf("CPU time: %13lf msec\n", executime);

    // Original
    size_t size = WIDTH * WIDTH * sizeof(int);
    int *Md, *Nd, *Pd, *Td;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipMalloc((void **)&Md, size);
    hipMemcpy(Md, M, size, hipMemcpyHostToDevice);
    hipMalloc((void **)&Nd, size);
    hipMemcpy(Nd, N, size, hipMemcpyHostToDevice);
    hipMalloc((void **)&Td, size);
    hipMalloc((void **)&Pd, size);

    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);
    dim3 dimGrid(BLOCKSPERGRID, BLOCKSPERGRID);

    hipEventRecord(start, 0);
    transpose<<<dimGrid, dimBlock>>>(Nd, Td);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime1, start, stop);
    printf("GPU transpose time: %13f msec\n", elapsedTime1);

    mat_mul<<<dimGrid, dimBlock>>>(Md, Td, Pd);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime2, start, stop);
    printf("GPU time: %13f msec\n", elapsedTime2);
    printf("GPU total time: %13f msec\n", elapsedTime1 + elapsedTime2);

    hipError_t cuda_err = hipGetLastError();
    if (hipSuccess != cuda_err) {
        printf("before kernel call: error = %s\n", hipGetErrorString (cuda_err));
        exit(1);
    }
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipMemcpy(P, Pd, size, hipMemcpyDeviceToHost);
    int pass = 1;
    for (int i = 0; i < WIDTH; ++i) {
        for (int j = 0; j < WIDTH; ++j) {
            if (MxN[i][j] != P[i][j]) {
                printf("MxN[%d][%d] = %d   P[%d][%d] = %d\n", i, j, MxN[i][j], i, j, P[i][j]);
                pass = 0;
                break;
            }
        }
    }
    printf("Test %s\n", (pass)?"PASSED":"FAILED");
    hipFree(Md);
    hipFree(Nd);
    hipFree(Pd);
    hipFree(Td);

    return 0;
}

__global__ void mat_mul(int *Md, int *Nd, int *Pd)
{
    int blockRow = blockIdx.y;
    int blockCol = blockIdx.x;
    
    int *Pd_sub = GetSubMatrix(Pd, blockRow, blockCol);
    
    int row = threadIdx.y;
    int col = threadIdx.x;
    
    int Pvalue = 0;
    
    __shared__ int Mds[TILE_WIDTH][TILE_WIDTH];
    __shared__ int Nds[TILE_WIDTH][TILE_WIDTH];
    
    for (int m = 0; m < (WIDTH / TILE_WIDTH); ++m) {
        int *Md_sub = GetSubMatrix(Md, blockRow, m);
        int *Nd_sub = GetSubMatrix(Nd, blockCol, m);
        
        Mds[row][col] = GetElement(Md_sub, row, col);
        Nds[row][col] = GetElement(Nd_sub, row, col);
        
        __syncthreads();
        
        for (int k = 0; k < TILE_WIDTH; ++k) {
            Pvalue += Mds[row][k] * Nds[col][k];
        }
        
        __syncthreads();
    }
    
    SetElement(Pd_sub, row, col, Pvalue);
}

__global__ void transpose(int *Nd, int *Td) {
    int xIndex = blockIdx.x * TILE_WIDTH + threadIdx.x;
    int yIndex = blockIdx.y * TILE_WIDTH + threadIdx.y;
    int index_in = xIndex + WIDTH * yIndex;
    int index_out = yIndex + WIDTH * xIndex;
    Td[index_out] = Nd[index_in];
}

__device__ int GetElement(int *matrix, int y, int x)
{
    return *(matrix + y * WIDTH + x);
}

__device__ void SetElement(int *matrix, int y, int x, int value)
{
    *(matrix + y * WIDTH + x) = value;
}

__device__ int *GetSubMatrix(int *matrix, int block_y, int block_x)
{
    return (matrix + block_y * TILE_WIDTH * WIDTH + block_x * TILE_WIDTH);
}
