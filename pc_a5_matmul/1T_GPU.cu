#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#define WIDTH 1024
#define THREADSPERBLOCK 16
#define BLOCKSPERGRID (WIDTH + THREADSPERBLOCK - 1) / THREADSPERBLOCK

int M[WIDTH][WIDTH] = {0};
int N[WIDTH][WIDTH] = {0};
int P[WIDTH][WIDTH] = {0};
int MxN[WIDTH][WIDTH] = {0};

__global__ void mat_mul(int *Md, int *Nd, int *Pd);
__global__ void transposeNaive(int *Nd, int *Td);

int main(int argc, char *argv[])
{
    float elapsedTime1;
    float elapsedTime2;

    for (int i = 0; i < WIDTH; ++i)
    {
        for (int j = 0; j < WIDTH; ++j)
        {
            M[i][j] = (int)(rand() % 255 + 1);
            N[i][j] = (int)(rand() % 255 + 1);
        }
    }

    struct timeval starttime, endtime;
    gettimeofday(&starttime, NULL);
    for (int i = 0; i < WIDTH; ++i)
    {
        for (int j = 0; j < WIDTH; ++j)
        {
            for (int k = 0; k < WIDTH; ++k)
            {
                MxN[i][j] += M[i][k] * N[k][j];
            }
        }
    }
    gettimeofday(&endtime, NULL);
    double executime;
    executime = (endtime.tv_sec - starttime.tv_sec) * 1000.0;
    executime += (endtime.tv_usec - starttime.tv_usec) / 1000.0;
    printf("CPU time: %13lf msec\n", executime);

    // Original
    size_t size = WIDTH * WIDTH * sizeof(int);
    int *Md, *Nd, *Pd, *Td;

    hipMalloc((void **)&Md, size);
    hipMemcpy(Md, M, size, hipMemcpyHostToDevice);
    hipMalloc((void **)&Nd, size);
    hipMemcpy(Nd, N, size, hipMemcpyHostToDevice);
    hipMalloc((void **)&Td, size);
    hipMalloc((void **)&Pd, size);

    dim3 dimGrid(BLOCKSPERGRID, BLOCKSPERGRID);
    dim3 dimBlock(THREADSPERBLOCK, THREADSPERBLOCK);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    transposeNaive<<<dimGrid, dimBlock>>>(Nd, Td);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime1, start, stop);
    printf("GPU transpose time: %13f msec\n", elapsedTime1);

    hipEventRecord(start, 0);
    mat_mul<<<dimGrid, dimBlock>>>(Md, Td, Pd);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime2, start, stop);
    printf("GPU time: %13f msec\n", elapsedTime2);
    printf("GPU total time: %13f msec\n", elapsedTime1 + elapsedTime2);
    hipError_t cuda_err = hipGetLastError();
    if (hipSuccess != cuda_err)
    {
        printf("before kernel call: error = %s\n", hipGetErrorString(cuda_err));
        exit(1);
    }
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipMemcpy(P, Pd, size, hipMemcpyDeviceToHost);
    int pass = 1;
    for (int i = 0; i < WIDTH; ++i)
    {
        for (int j = 0; j < WIDTH; ++j)
        {
            if (MxN[i][j] != P[i][j])
            {
                printf("MxN[%d][%d] = %d   P[%d][%d] = %d\n", i, j, MxN[i][j], i, j, P[i][j]);
                pass = 0;
                break;
            }
        }
    }
    printf("Test %s\n", (pass) ? "PASSED" : "FAILED");
    hipFree(Md);
    hipFree(Nd);
    hipFree(Td);
    hipFree(Pd);

    return 0;
}

__global__ void transposeNaive(int *Nd, int *Td)
{
    int x, y;
    int tx = threadIdx.x;
    int ty = blockDim.x * threadIdx.y;
    int bx = blockDim.x * blockDim.y * blockIdx.x;
    int by = gridDim.x * (blockDim.x * blockDim.y) * blockIdx.y;
    int tid = bx + by + tx + ty;
    while (tid < WIDTH * WIDTH)
    {
        x = tid % WIDTH;
        y = tid / WIDTH;
        Td[x * WIDTH + y] = Nd[y * WIDTH + x];
        tid = tid + gridDim.x * gridDim.y * blockDim.x * blockDim.y;
    }
}

__global__ void mat_mul(int *Md, int *Nd, int *Pd)
{
    int x, y;
    int Pvalue;
    int tx = threadIdx.x;
    int ty = blockDim.x * threadIdx.y;
    int bx = blockDim.x * blockDim.y * blockIdx.x;
    int by = gridDim.x * (blockDim.x * blockDim.y) * blockIdx.y;
    int tid = bx + by + tx + ty;

    while (tid < WIDTH * WIDTH)
    {
        x = tid % WIDTH;
        y = tid / WIDTH;
        Pvalue = 0;
        for (int k = 0; k < WIDTH; ++k)
        {
            int Melement = *(Md + y * WIDTH + k);
            int Nelement = *(Nd + x * WIDTH + k);
            Pvalue += Melement * Nelement;
        }
        *(Pd + y * WIDTH + x) = Pvalue;
        tid = tid + gridDim.x * gridDim.y * blockDim.x * blockDim.y;
    }
}
